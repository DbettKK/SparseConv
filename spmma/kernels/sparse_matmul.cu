#include "hip/hip_runtime.h"
//
// Created by dbettkk on 2022/3/30.
//
#include"sparse_matmul.cuh"

void
spmma_matmul(const half *inputA, const half *inputB, int inputM, int inputK, int inputN, bool isValid, half *outputD,
             MatrixParam *retParam) {

    auto time_all = new CudaTime();
    time_all->initAndStart();

    int m = inputM % 8 ? inputM + 8 - inputM % 8 : inputM;
    int k = inputK % 16 ? inputK + 16 - inputK % 16 : inputK;
    int n = inputN % 8 ? inputN + 8 - inputN % 8 : inputN;

    size_t A_size = m * k * sizeof(half);
    size_t B_size = k * n * sizeof(half);
    size_t C_size = m * n * sizeof(half);
    // device
    half *dA, *dB, *dC, *dD, *dA_compressed;
    int *d_valid;
    int *is_valid = (int *)malloc(sizeof(int));
    CHECK_CUDA(hipMalloc((void **) &dA, A_size))
    CHECK_CUDA(hipMalloc((void **) &dB, B_size))
    CHECK_CUDA(hipMalloc((void **) &dC, C_size))
    CHECK_CUDA(hipMalloc((void **) &d_valid, sizeof(d_valid)))
    CHECK_CUDA(hipMemset(dC, 0, C_size))
    dD = dC;

    // padding to match mma.sp
    padMatrix<half>(inputA, inputM, inputK, dA, m, k);
    padMatrix<half>(inputB, inputK, inputN, dB, k, n);

//    if (!check_sparse(dA, m, k)) printf("not match\n");
//    else printf("match\n");

    // Leading dimension 如果行优先则代表列数
    int lda = k, ldb = n, ldc = n;
    auto opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    auto opB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    auto order = HIPSPARSE_ORDER_ROW; // hipsparseOrder_t
    auto type = HIP_R_16F;
    auto compute_type = CUSPARSE_COMPUTE_16F;
    float alpha = 1.0f;
    float beta = 0.0f;
    unsigned alignment = 16;

    //--------------------------------------------------------------------------

    cusparseLtHandle_t handle;
    cusparseLtMatDescriptor_t matA, matB, matC;
    cusparseLtMatmulDescriptor_t matmul;
    cusparseLtMatmulAlgSelection_t alg_sel;
    cusparseLtMatmulPlan_t plan;
    hipStream_t stream = nullptr;
    CHECK_CUSPARSE(cusparseLtInit(&handle))
    // matrix descriptor initialization
    CHECK_CUSPARSE(cusparseLtStructuredDescriptorInit(&handle, &matA, m, k, lda, alignment, type, order,
                                                      CUSPARSELT_SPARSITY_50_PERCENT))
    CHECK_CUSPARSE(cusparseLtDenseDescriptorInit(&handle, &matB, k, n, ldb, alignment, type, order))
    CHECK_CUSPARSE(cusparseLtDenseDescriptorInit(&handle, &matC, m, n, ldc, alignment, type, order))
    // matmul, algorithm selection, and plan initialization
    CHECK_CUSPARSE(cusparseLtMatmulDescriptorInit(&handle, &matmul, opA, opB, &matA, &matB, &matC, &matC, compute_type))
    CHECK_CUSPARSE(cusparseLtMatmulAlgSelectionInit(&handle, &alg_sel, &matmul, CUSPARSELT_MATMUL_ALG_DEFAULT))
    int alg = 0;    // 算法
    CHECK_CUSPARSE(
            cusparseLtMatmulAlgSetAttribute(&handle, &alg_sel, CUSPARSELT_MATMUL_ALG_CONFIG_ID, &alg, sizeof(alg)))

    size_t workspace_size, compressed_size;
    CHECK_CUSPARSE(cusparseLtMatmulGetWorkspace(&handle, &alg_sel, &workspace_size))

    CHECK_CUSPARSE(cusparseLtMatmulPlanInit(&handle, &plan, &matmul, &alg_sel, workspace_size))
    //--------------------------------------------------------------------------
    // Prune and Compress
    if (!isValid) {
        // 不符合条件 需要进行剪枝
        //int is_valid = 0;
        CHECK_CUSPARSE(cusparseLtSpMMAPruneCheck(&handle, &matmul, dA, d_valid, stream))
        CHECK_CUDA(hipMemcpyAsync(is_valid, d_valid, sizeof(d_valid), hipMemcpyDeviceToHost, stream))
        CHECK_CUDA(hipStreamSynchronize(stream))
        if (*is_valid == 1) {
            std::printf("!!!! The matrix need to be pruned. valid: %d\n", *is_valid);
            CHECK_CUSPARSE(cusparseLtSpMMAPrune(&handle, &matmul, dA, dA, CUSPARSELT_PRUNE_SPMMA_TILE, stream))
        }
    }
    // 符合条件 不用判断 直接compress即可
    CHECK_CUSPARSE(cusparseLtSpMMACompressedSize(&handle, &plan, &compressed_size))
    CHECK_CUDA(hipMalloc((void **) &dA_compressed, compressed_size))
    CHECK_CUSPARSE(cusparseLtSpMMACompress(&handle, &plan, dA, dA_compressed, stream))
    // 对compress后的进行拆分

//    half *data_cmpr = new half[compressed_size / sizeof(half)]; // data部分
//    int *index = (int *) malloc(compressed_size / 2);
//    size_t index_t = compressed_size - m * k / 2 * sizeof(half);
//    hipMemcpy(data_cmpr, dA_compressed, m * k / 2 * sizeof(half), hipMemcpyDeviceToHost);
//    hipMemcpy(index, dA_compressed + m * k / 2, index_t, hipMemcpyDeviceToHost);
//    printf("cmpr_size: %llu\n", compressed_size - m * k / 2 * sizeof(half));
//    printf("my cmpr_size: %zu\n", get_cmpr_size(m, k));
//    printf("m * k: %d\n", m * k);
//    printf("data_cmpr:\n");
//    for (int i = 0; i < m; i++) {
//        for (int j = 0; j < k / 2; j++) {
//            printf("%d ", __half2int_rz(data_cmpr[i * k / 2 + j]));
//        }
//        printf("\n");
//    }
//
//    printf("index:\n");
//    printf("max_index: %llu\n", index_t / sizeof(int));
//    for (int i = 0; i < index_t / sizeof(int); i++) {
//        if (index[i] == -286331154) continue;
//        printf("%d: %d:", i, index[i]);
//        decimal2binary(index[i], 32);
//        printf("\n");
//    }
    //--------------------------------------------------------------------------

    //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
    // Search the best kernel
    void*         d_workspace = nullptr;
    int           num_streams = 0;
    hipStream_t* streams     = nullptr;
    /*
    int alg_id;
    CHECK_CUSPARSE( cusparseLtMatmulAlgGetAttribute(&handle, &alg_sel, CUSPARSELT_MATMUL_ALG_CONFIG_ID, &alg_id, sizeof(alg_id)) )
    printf("best alg: %d\n", alg_id);
    */
    //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
    // Perform the matrix multiplication

    // time
    auto time = new CudaTime();
    time->initAndStart();

    CHECK_CUSPARSE(cusparseLtMatmul(&handle, &plan, &alpha, dA_compressed, dB, &beta, dC, dD, d_workspace, streams,
                                    num_streams))

    float totalTime = time->endAndGetTime();
    //printf("cusparselt calculate took %fms\t", totalTime);
    printf("%f\t", totalTime);

    //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~

    // destroy plan and handle
    CHECK_CUSPARSE(cusparseLtMatDescriptorDestroy(&matA))
    CHECK_CUSPARSE(cusparseLtMatDescriptorDestroy(&matB))
    CHECK_CUSPARSE(cusparseLtMatDescriptorDestroy(&matC))
    CHECK_CUSPARSE(cusparseLtMatmulPlanDestroy(&plan))
    CHECK_CUSPARSE(cusparseLtDestroy(&handle))
    //--------------------------------------------------------------------------

    // padding后的fix
    CHECK_CUDA(hipMemcpy2D(outputD, inputN * sizeof(half), dD, n * sizeof(half), inputN * sizeof(half), inputM,
                            hipMemcpyDeviceToDevice))

    if (retParam != nullptr) {
        retParam->initIfNull();
        // 此时的dC已经不是全0了
        half *tmpC;
        CHECK_CUDA(hipMalloc((void **) &tmpC, m * n * sizeof(half)))
        CHECK_CUDA(hipMemset(tmpC, 0, m * n * sizeof(half)))
        retParam->copyFromDevice(dA, dB, tmpC, dD, m, k, n);
        CHECK_CUDA(hipFree(tmpC))
    }

    CHECK_CUDA( hipFree(dA_compressed) )
    CHECK_CUDA( hipFree(dA) )
    CHECK_CUDA( hipFree(dB) )
    CHECK_CUDA( hipFree(dC) )
    CHECK_CUDA( hipFree(d_valid) )

    float all_time = time_all->endAndGetTime();
    //printf("spmma all took %fms\n", all_time);

    ofstream out("../data/spmma_time.txt", ios::app);
    out << "spmma_matmul: " << all_time << "ms\n";
    out.close();


}


