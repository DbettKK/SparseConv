//
// Created by dbettkk on 2022/3/30.
//

#include "CudaTime.cuh"

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at %s line %d with error: %s (%d)\n",          \
               __FILE__, __LINE__, hipGetErrorString(status), status);        \
        return;                                                                \
    }                                                                          \
}

#define CHECK_CUDA_NO_RET(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at %s line %d with error: %s (%d)\n",          \
               __FILE__, __LINE__, hipGetErrorString(status), status);        \
        return 0;                                                              \
    }                                                                          \
}

void CudaTime::init() {
    CHECK_CUDA( hipEventCreateWithFlags(&startTime, hipEventBlockingSync) )
    CHECK_CUDA( hipEventCreateWithFlags(&endTime, hipEventBlockingSync) )
    CHECK_CUDA( hipEventCreate(&startTime) )
    CHECK_CUDA( hipEventCreate(&endTime) )
}

void CudaTime::start() {
    CHECK_CUDA( hipEventRecord(startTime) )
}

void CudaTime::end() {
    CHECK_CUDA( hipEventRecord(endTime) )
}

float CudaTime::getTime() {
    float totalTime;
    //hipEventQuery(endTime);
    CHECK_CUDA_NO_RET( hipEventSynchronize(endTime) )
    CHECK_CUDA_NO_RET( hipEventElapsedTime(&totalTime, startTime, endTime) )
    return totalTime;
}

void CudaTime::destroy() {
    CHECK_CUDA( hipEventDestroy(startTime) )
    CHECK_CUDA( hipEventDestroy(endTime) )
}

void CudaTime::initAndStart() {
    init();
    start();
}

float CudaTime::endAndGetTime() {
    end();
    float totalTime = getTime();
    destroy();
    return totalTime;
}

void CudaTime::endAndPrintTime(const std::string &msg) {
    printf("%s %fms\n", msg.c_str(), endAndGetTime());
}

void CudaTime::endAndExportTimeToFile(const std::string path, const std::string prefix_msg) {
    float time = endAndGetTime();
    std::ofstream out(path, std::ios::app);
    out << prefix_msg << time << "ms\n";
    out.close();
}
