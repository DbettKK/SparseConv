//
// Created by dbettkk on 2022/3/30.
//

#include "CudaTime.cuh"

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at %s line %d with error: %s (%d)\n",          \
               __FILE__, __LINE__, hipGetErrorString(status), status);        \
        return;                                                                \
    }                                                                          \
}

#define CHECK_CUDA_NO_RET(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at %s line %d with error: %s (%d)\n",          \
               __FILE__, __LINE__, hipGetErrorString(status), status);        \
        return 0;                                                              \
    }                                                                          \
}

void CudaTime::init() {
    CHECK_CUDA( hipEventCreateWithFlags(&startTime, hipEventBlockingSync) )
    CHECK_CUDA( hipEventCreateWithFlags(&endTime, hipEventBlockingSync) )
    CHECK_CUDA( hipEventCreate(&startTime) )
    CHECK_CUDA( hipEventCreate(&endTime) )
}

void CudaTime::start() {
    CHECK_CUDA( hipEventRecord(startTime) )
}

void CudaTime::end() {
    CHECK_CUDA( hipEventRecord(endTime) )
}

float CudaTime::getTime() {
    float totalTime;
    CHECK_CUDA_NO_RET( hipEventSynchronize(endTime) )
    CHECK_CUDA_NO_RET( hipEventElapsedTime(&totalTime, startTime, endTime) )
    return totalTime;
}

void CudaTime::destroy() {
    CHECK_CUDA( hipEventDestroy(startTime) )
    CHECK_CUDA( hipEventDestroy(endTime) )
}

void CudaTime::initAndStart() {
    init();
    start();
}

float CudaTime::endAndGetTime() {
    end();
    float totalTime = getTime();
    destroy();
    return totalTime;
}

void CudaTime::endAndPrintTime(const std::string &msg) {
    end();
    float totalTime = getTime();
    destroy();
    printf("%s %fms\n", msg.c_str(), totalTime);
}
