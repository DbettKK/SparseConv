#include "hip/hip_runtime.h"
//
// Created by dbettkk on 2022/7/23.
//

#include "MatrixHalf.cuh"

MatrixHalf::MatrixHalf(half *matrix, int batch, int row, int col) : matrix(matrix), batch(batch), row(row), col(col) {}

MatrixHalf::MatrixHalf(int batch, int row, int col, bool is_device) : batch(batch), row(row), col(col) {
    if (!is_device) this->matrix = new half[batch * row * col];
    else CHECK_CUDA(hipMalloc(&this->matrix, sizeof(half) * batch * row * col))
}

MatrixHalf::MatrixHalf(int batch, int row, int col, bool is_device, half init) : batch(batch), row(row), col(col) {
    half *tmp = new half[batch * row * col];
    for (int i = 0; i < batch * row * col; i++) tmp[i] = init;
    if (is_device) {
        CHECK_CUDA(hipMalloc(&matrix, sizeof(half) * row * col * batch));
        CHECK_CUDA(hipMemcpy(matrix, tmp, sizeof(half) * row * col * batch, hipMemcpyHostToDevice));
        delete[] tmp;
    } else {
        matrix = tmp;
    }
}

half *MatrixHalf::getMatrix() const {
    return matrix;
}

void MatrixHalf::setMatrix(half *matrix) {
    MatrixHalf::matrix = matrix;
}

int MatrixHalf::getBatch() const {
    return batch;
}

void MatrixHalf::setBatch(int batch) {
    MatrixHalf::batch = batch;
}

int MatrixHalf::getRow() const {
    return row;
}

void MatrixHalf::setRow(int row) {
    MatrixHalf::row = row;
}

int MatrixHalf::getCol() const {
    return col;
}

void MatrixHalf::setCol(int col) {
    MatrixHalf::col = col;
}

void MatrixHalf::gemm(MatrixHalf *item, MatrixHalf *out) {
    if (this->batch != out->getBatch()) {
        printf("source batch_size should be equaled to the target batch_size!");
        return;
    }
    cublas_gemm_device(this->matrix, item->matrix, this->row, this->col, item->col, out->matrix);
    //sparse_mma_gemm_device(this->matrix, item->matrix, this->row, this->col, item->col, true, out->matrix);
    //dim3 grid(16, 16);
    //dim3 block(32, 32);
    //gemm_simple<<<grid, block>>>(this->matrix, item->matrix, row, col, item->col, out->matrix);
}

void MatrixHalf::gemm_batches(MatrixHalf *item, MatrixHalf *out, bool is_single_batch) {
    // is_single_batch: item的batch数是否为1
    if (is_single_batch) {
        for (int i = 0; i < batch; i++) {
            cublas_gemm_device(matrix + i * row * col, item->matrix, row, col, item->col,
                               out->matrix + i * out->row * out->col);
        }
    } else {
        for (int i = 0; i < batch; i++) {
            cublas_gemm_device(matrix + i * row * col, item->matrix + i * item->row * item->col,
                               row, col, item->col, out->matrix + i * out->row * out->col);
        }
    }
}

int MatrixHalf::getSize() const {
    return this->batch * this->col * this->row;
}

void MatrixHalf::reshape(MatrixHalf *out, int heads) const {
    dim3 block(this->batch, heads);
    dim3 thread(this->row, this->col / heads);
    reshape_multi_head<<<block, thread>>>(this->matrix, out->matrix, this->row, this->col, heads);
}

void MatrixHalf::transpose(MatrixHalf *out) {
    dim3 thread(this->row, this->col);
    transpose_half<<<1, thread>>>(this->matrix, out->matrix, this->row, this->col);
}

void MatrixHalf::softmax() {
    softmax_half<<<this->col, this->row>>>(this->matrix, this->row, this->col);
}

void MatrixHalf::print(const std::string& msg, bool is_device) {
    std::cout << msg << std::endl;
    if (is_device) {
        half *tmp = new half[row * col];
        CHECK_CUDA(hipMemcpy(tmp, matrix, sizeof(half) * row * col, hipMemcpyDeviceToHost));
        for (int i = 0; i < row; i++) {
            for (int j = 0; j < col; j++) {
                printf("%.2f ", __half2float(tmp[i * col + j]));
            }
            printf("\n");
        }
        delete[] tmp;
    } else {
        for (int i = 0; i < row; i++) {
            for (int j = 0; j < col; j++) {
                printf("%.2f ", __half2float(matrix[i * col + j]));
            }
            printf("\n");
        }
    }

}

void MatrixHalf::free_matrix() {
    hipError_t st = hipFree(matrix);
    if (st != hipSuccess) {
        printf("CUDA API failed at %s line %d with error: %s (%d)\n",
               __FILE__, __LINE__, hipGetErrorString(st), st);
        return;
    }
    //CHECK_CUDA(hipFree(matrix))
}

 void MatrixHalf::print_device(half *item, int row, int col) {
    // std::cout << msg << std::endl;
    half *tmp = new half[row * col];
    CHECK_CUDA(hipMemcpy(tmp, item, sizeof(half) * row * col, hipMemcpyDeviceToHost));
    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            printf("%.2f ", __half2float(tmp[i * col + j]));
        }
        printf("\n");
    }
    delete[] tmp;
}

void MatrixHalf::relu() {
    dim3 grid(row / 32 + 1, col / 32 + 1);
    dim3 block(32, 32);
    relu_half<<<grid, block>>>(matrix, row, col);
}

void MatrixHalf::addMatrix(MatrixHalf *add, MatrixHalf *out) {
    if (this->row != add->row || this->col != add->col) {
        printf("Error: matrix add need to be the same shape!\n");
        return;
    }
    matrix_add<<<batch, row * col>>>(matrix, add->matrix, out->matrix, batch * row * col);
}

void MatrixHalf::copyTo(MatrixHalf *out) {
    hipMemcpy(out->getMatrix(), matrix, sizeof(half) * out->getSize(), hipMemcpyDeviceToDevice);
}



