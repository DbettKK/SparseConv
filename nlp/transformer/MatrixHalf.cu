#include "hip/hip_runtime.h"
//
// Created by dbettkk on 2022/7/23.
//

#include "MatrixHalf.cuh"

MatrixHalf::MatrixHalf(half *matrix, int batch, int row, int col) : matrix(matrix), batch(batch), row(row), col(col) {}

MatrixHalf::MatrixHalf(int batch, int row, int col, bool is_device) : batch(batch), row(row), col(col) {
    if (!is_device) this->matrix = new half[batch * row * col];
    else CHECK_CUDA(hipMalloc(&this->matrix, sizeof(half) * batch * row * col))
}

MatrixHalf::MatrixHalf(int batch, int row, int col, bool is_device, half init) : batch(batch), row(row), col(col) {
    half *tmp = new half[batch * row * col];
    for (int i = 0; i < batch * row * col; i++) tmp[i] = init;
    if (is_device) {
        CHECK_CUDA(hipMalloc(&matrix, sizeof(half) * row * col * batch));
        CHECK_CUDA(hipMemcpy(matrix, tmp, sizeof(half) * row * col * batch, hipMemcpyHostToDevice));
        delete[] tmp;
    } else {
        matrix = tmp;
    }
}

half *MatrixHalf::getMatrix() const {
    return matrix;
}

void MatrixHalf::setMatrix(half *matrix) {
    MatrixHalf::matrix = matrix;
}

int MatrixHalf::getBatch() const {
    return batch;
}

void MatrixHalf::setBatch(int batch) {
    MatrixHalf::batch = batch;
}

int MatrixHalf::getRow() const {
    return row;
}

void MatrixHalf::setRow(int row) {
    MatrixHalf::row = row;
}

int MatrixHalf::getCol() const {
    return col;
}

void MatrixHalf::setCol(int col) {
    MatrixHalf::col = col;
}

void MatrixHalf::gemm(MatrixHalf *item, MatrixHalf *out) {
    cublas_gemm_device(this->matrix, item->matrix, this->row, this->col, item->col, out->matrix);
    //sparse_mma_gemm_device(this->matrix, item->matrix, this->row, this->col, item->col, true, out->matrix);
    //dim3 grid(16, 16);
    //dim3 block(32, 32);
    //gemm_simple<<<grid, block>>>(this->matrix, item->matrix, row, col, item->col, out->matrix);
}

int MatrixHalf::getSize() const {
    return this->batch * this->col * this->row;
}

void MatrixHalf::reshape(MatrixHalf *out, int heads) const {
    dim3 thread(this->row, this->col / heads);
    reshape_multi_head<<<heads, thread>>>(this->matrix, out->matrix, this->row, this->col, heads);
}

void MatrixHalf::transpose(MatrixHalf *out) {
    dim3 thread(this->row, this->col);
    transpose_half<<<1, thread>>>(this->matrix, out->matrix, this->row, this->col);
}

void MatrixHalf::softmax() {
    softmax_half<<<this->col, this->row>>>(this->matrix, this->row, this->col);
}

void MatrixHalf::print(const std::string& msg, bool is_device) {
    std::cout << msg << std::endl;
    if (is_device) {
        half *tmp = new half[row * col];
        CHECK_CUDA(hipMemcpy(tmp, matrix, sizeof(half) * row * col, hipMemcpyDeviceToHost));
        for (int i = 0; i < row; i++) {
            for (int j = 0; j < col; j++) {
                printf("%.2f ", __half2float(tmp[i * col + j]));
            }
            printf("\n");
        }
        delete[] tmp;
    } else {
        for (int i = 0; i < row; i++) {
            for (int j = 0; j < col; j++) {
                printf("%.2f ", __half2float(matrix[i * col + j]));
            }
            printf("\n");
        }
    }

}



