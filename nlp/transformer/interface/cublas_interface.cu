#include "hip/hip_runtime.h"
//
// Created by dbettkk on 2022/8/14.
//
#include "cublas_interface.cuh"

void cublas_gemm_device(const half *d_A, const half *d_B, int inputM, int inputK, int inputN, half *output) {
    // 因为为列存储，为了方便，设置转置
    hipblasHandle_t cublasH = nullptr;

    const int m = inputM;
    const int n = inputN;
    const int k = inputK;
    const int lda = k; // 因为转置了 因此ld代表列数
    const int ldb = n;
    const int ldc = m; // c的ld都是m

    const half alpha = 1.0;
    const half beta = 0.0;

    half *d_C = nullptr;

    hipblasOperation_t transa = HIPBLAS_OP_T;
    hipblasOperation_t transb = HIPBLAS_OP_T;

    /* step 1: create cublas handle, bind a stream */
    CHECK_CUBLAS( hipblasCreate(&cublasH) );

    /* step 2: copy data to device */
    CHECK_CUDA( hipMalloc(&d_C, sizeof(half) * m * n) );

    /* step 3: compute */
    CHECK_CUBLAS( hipblasHgemm(cublasH, transa, transb, m, n, k, &alpha, d_A, lda, d_B, ldb, &beta, d_C, ldc) );

    // transpose
    dim3 grid(m / 32 + 1, n / 32 + 1);
    dim3 block(32, 32);
    transpose_half<<<grid, block>>>(d_C, output, m, n);

    /* step 4: copy data to host */
    //CHECK_CUDA( hipMemcpyAsync(output, d_C, sizeof(half) * m * n, hipMemcpyDeviceToDevice, stream));

    /* free resources */
    CHECK_CUDA( hipFree(d_C) );
    CHECK_CUBLAS( hipblasDestroy(cublasH) );

}

void cublas_gemm_device_scale(const half *d_A, const half *d_B, int inputM, int inputK, int inputN, float scale, half *output) {
    // 因为为列存储，为了方便，设置转置
    hipblasHandle_t cublasH = nullptr;

    const int m = inputM;
    const int n = inputN;
    const int k = inputK;
    const int lda = k; // 因为转置了 因此ld代表列数
    const int ldb = n;
    const int ldc = m; // c的ld都是m

    const half alpha = 1.0;
    const half beta = 0.0;

    half *d_C = nullptr;

    hipblasOperation_t transa = HIPBLAS_OP_T;
    hipblasOperation_t transb = HIPBLAS_OP_T;

    /* step 1: create cublas handle, bind a stream */
    CHECK_CUBLAS( hipblasCreate(&cublasH) );

    /* step 2: copy data to device */
    CHECK_CUDA( hipMalloc(&d_C, sizeof(half) * m * n) );

    /* step 3: compute */
    CHECK_CUBLAS( hipblasHgemm(cublasH, transa, transb, m, n, k, &alpha, d_A, lda, d_B, ldb, &beta, d_C, ldc) );

    CHECK_CUBLAS( hipblasScalEx(cublasH, m * n, &scale, HIP_R_32F, d_C, HIP_R_16F, 1, HIP_R_32F) )

    // transpose
    dim3 grid(m / 32 + 1, n / 32 + 1);
    dim3 block(32, 32);
    transpose_half<<<grid, block>>>(d_C, output, m, n);

    /* step 4: copy data to host */
    //CHECK_CUDA( hipMemcpyAsync(output, d_C, sizeof(half) * m * n, hipMemcpyDeviceToDevice, stream));

    /* free resources */
    CHECK_CUDA( hipFree(d_C) );
    CHECK_CUBLAS( hipblasDestroy(cublasH) );

}

void cublas_gemm_batches_device(half *d_A, half *d_B, int batch, int inputM, int inputK, int inputN,
                                bool isSingleBatch, half *output) {
    const int m = inputM;
    const int n = inputN;
    const int k = inputK;
    const int lda = k; // 因为转置了 因此ld代表列数
    const int ldb = n;
    const int ldc = m; // c的ld都是m

    const half alpha = 1.0;
    const half beta = 0.0;

    half *d_C = nullptr;

    hipblasOperation_t transa = HIPBLAS_OP_T;
    hipblasOperation_t transb = HIPBLAS_OP_T;

    /* step 1: create cublas handle, bind a stream */
    hipblasHandle_t handle = nullptr;
    CHECK_CUBLAS( hipblasCreate(&handle) );

    /* step 2: copy data to device */
    CHECK_CUDA( hipMalloc(&d_C, sizeof(half) * batch * m * n) );
    CHECK_CUDA( hipMemset(d_C, 0, sizeof(half) * batch * m * n) );

    /* step 3: compute */
    half **dArrA, **dArrB, **dArrC;
    half *arrA[64], *arrB[64], *arrC[64];
    if (isSingleBatch) {
        for (int i = 0; i < batch; i++) {
            half *tmpB;
            CHECK_CUDA(hipMalloc(&tmpB, sizeof(half) * k * n))
            CHECK_CUDA(hipMemcpy(tmpB, d_B, sizeof(half) * k * n, hipMemcpyDeviceToDevice))
            arrA[i] = d_A + i * m * k;
            arrB[i] = tmpB;
            arrC[i] = d_C + i * m * n;
        }
    } else {
        for (int i = 0; i < batch; i++) {
            arrA[i] = d_A + i * m * k;
            arrB[i] = d_B + i * n * k;
            arrC[i] = d_C + i * m * n;
        }
    }
    CHECK_CUDA(hipMalloc(&dArrA, sizeof(half*) * batch))
    CHECK_CUDA(hipMalloc(&dArrB, sizeof(half*) * batch))
    CHECK_CUDA(hipMalloc(&dArrC, sizeof(half*) * batch))
    CHECK_CUDA(hipMemcpy(dArrA, arrA, sizeof(half*) * batch, hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(dArrB, arrB, sizeof(half*) * batch, hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(dArrC, arrC, sizeof(half*) * batch, hipMemcpyHostToDevice))

    CHECK_CUBLAS(hipblasHgemmBatched(handle, transa, transb, m, n, k, &alpha, dArrA, lda, dArrB, ldb, &beta, dArrC, ldc, batch))

    CHECK_CUDA(hipMemcpy(arrC, dArrC, sizeof(half*) * batch, hipMemcpyDeviceToHost))

    for (int i = 0; i < batch; i++) {
        CHECK_CUDA(hipMemcpy(output + i * m * n, arrC[i], sizeof(half) * m * n, hipMemcpyDeviceToDevice));
    }
//    for (int i = 0; i < batch; i++) {
//        half *c_out = new half[m * n];
//        hipMemcpy(c_out, arrC[i], sizeof(half) * m * k, hipMemcpyDeviceToHost);
//        //hipMemcpy(c_out, dArrC + i, sizeof(half) * m * n, hipMemcpyDeviceToHost);
//        for (int j = 0; j < m; j++) {
//            for (int v = 0; v < n; v++) {
//                printf("%d ", __half2int_rz(c_out[j * n + v]));
//            }
//            printf("\n");
//        }
//    }
    /* free resources */
    CHECK_CUDA( hipFree(d_C) );
    CHECK_CUDA( hipFree(dArrA) );
    CHECK_CUDA( hipFree(dArrB) );
    CHECK_CUDA( hipFree(dArrC) );
    CHECK_CUBLAS( hipblasDestroy(handle) );

}

void cublas_gemm_batches_device_v2(half *d_A, half *d_B, int batch, int inputM, int inputK, int inputN, bool isSingleBatch, half *output) {
    // 因为为列存储，为了方便，设置转置
    hipblasHandle_t cublasH = nullptr;

    const int m = inputM;
    const int n = inputN;
    const int k = inputK;
    const int lda = k; // 因为转置了 因此ld代表列数
    const int ldb = n;
    const int ldc = m; // c的ld都是m

    const half alpha = 1.0;
    const half beta = 0.0;
    int64_t strideA = m * k, strideB = isSingleBatch ? 0 : k * n, strideC = m * n;

    half *d_C = nullptr;

    hipblasOperation_t transa = HIPBLAS_OP_T;
    hipblasOperation_t transb = HIPBLAS_OP_T;

    /* step 1: create cublas handle, bind a stream */
    CHECK_CUBLAS( hipblasCreate(&cublasH) );

    /* step 2: copy data to device */
    CHECK_CUDA( hipMalloc(&d_C, sizeof(half) * batch * m * n) );

    /* step 3: compute */
    CHECK_CUBLAS( hipblasHgemmStridedBatched(cublasH, transa, transb, m, n, k, &alpha, d_A, lda, strideA,
                                            d_B, ldb, strideB, &beta, d_C, ldc, strideC, batch) );

    // transpose
    dim3 grid(batch / 32 + 1, m, n);
    transpose_batches<<<grid, 32>>>(d_C, output, batch, m, n);

    /* step 4: copy data to host */
    //CHECK_CUDA( hipMemcpyAsync(output, d_C, sizeof(half) * m * n, hipMemcpyDeviceToDevice, stream));

    /* free resources */
    CHECK_CUDA( hipFree(d_C) );
    CHECK_CUBLAS( hipblasDestroy(cublasH) );
}

void cublas_gemm_batches_scale_device(half *d_A, half *d_B, int batch, int inputM, int inputK, int inputN,
                                      float scale, half *output) {
    const int m = inputM;
    const int n = inputN;
    const int k = inputK;
    const int lda = k; // 因为转置了 因此ld代表列数
    const int ldb = n;
    const int ldc = m; // c的ld都是m

    const half alpha = 1.0;
    const half beta = 0.0;

    half *d_C = nullptr;

    hipblasOperation_t transa = HIPBLAS_OP_T;
    hipblasOperation_t transb = HIPBLAS_OP_T;

    /* step 1: create cublas handle, bind a stream */
    hipblasHandle_t handle = nullptr;
    CHECK_CUBLAS( hipblasCreate(&handle) );

    /* step 2: copy data to device */
    CHECK_CUDA( hipMalloc(&d_C, sizeof(half) * batch * m * n) );
    CHECK_CUDA( hipMemset(d_C, 0, sizeof(half) * batch * m * n) );

    /* step 3: compute */
    half **dArrA, **dArrB, **dArrC;
    half *arrA[64], *arrB[64], *arrC[64];
    for (int i = 0; i < batch; i++) {
        arrA[i] = d_A + i * m * k;
        arrB[i] = d_B + i * n * k;
        arrC[i] = d_C + i * m * n;
    }
    CHECK_CUDA(hipMalloc(&dArrA, sizeof(half*) * batch))
    CHECK_CUDA(hipMalloc(&dArrB, sizeof(half*) * batch))
    CHECK_CUDA(hipMalloc(&dArrC, sizeof(half*) * batch))
    CHECK_CUDA(hipMemcpy(dArrA, arrA, sizeof(half*) * batch, hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(dArrB, arrB, sizeof(half*) * batch, hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(dArrC, arrC, sizeof(half*) * batch, hipMemcpyHostToDevice))

    CHECK_CUBLAS(hipblasHgemmBatched(handle, transa, transb, m, n, k, &alpha, dArrA, lda, dArrB, ldb, &beta, dArrC, ldc, batch))

    CHECK_CUDA(hipMemcpy(arrC, dArrC, sizeof(half*) * batch, hipMemcpyDeviceToHost))

    half *tmp_out;
    CHECK_CUDA(hipMalloc(&tmp_out, sizeof(half) * batch * m * n))

    for (int i = 0; i < batch; i++) {
        CHECK_CUDA(hipMemcpy(tmp_out + i * m * n, arrC[i], sizeof(half) * m * n, hipMemcpyDeviceToDevice));
    }

    CHECK_CUBLAS( hipblasScalEx(handle, batch * m * n, &scale, HIP_R_32F, tmp_out, HIP_R_16F, 1, HIP_R_32F) )

    dim3 grid(batch / 32 + 1, m, n);
    transpose_batches<<<grid, 32>>>(tmp_out, output, batch, m, n);

//    for (int i = 0; i < batch; i++) {
//        half *c_out = new half[m * n];
//        hipMemcpy(c_out, arrC[i], sizeof(half) * m * k, hipMemcpyDeviceToHost);
//        //hipMemcpy(c_out, dArrC + i, sizeof(half) * m * n, hipMemcpyDeviceToHost);
//        for (int j = 0; j < m; j++) {
//            for (int v = 0; v < n; v++) {
//                printf("%d ", __half2int_rz(c_out[j * n + v]));
//            }
//            printf("\n");
//        }
//    }
    /* free resources */
    CHECK_CUDA( hipFree(d_C) );
    CHECK_CUDA( hipFree(dArrA) );
    CHECK_CUDA( hipFree(dArrB) );
    CHECK_CUDA( hipFree(dArrC) );
    CHECK_CUBLAS( hipblasDestroy(handle) );

}

void cublas_gemm_batches_scale_device_v2(half *d_A, half *d_B, int batch, int inputM, int inputK, int inputN,
                                         float scale, half *output) {
    // 因为为列存储，为了方便，设置转置
    hipblasHandle_t cublasH = nullptr;

    const int m = inputM;
    const int n = inputN;
    const int k = inputK;
    const int lda = k; // 因为转置了 因此ld代表列数
    const int ldb = n;
    const int ldc = m; // c的ld都是m

    const half alpha = 1.0;
    const half beta = 0.0;
    int64_t strideA = m * k, strideB = k * n, strideC = m * n;

    half *d_C = nullptr;

    hipblasOperation_t transa = HIPBLAS_OP_T;
    hipblasOperation_t transb = HIPBLAS_OP_T;

    /* step 1: create cublas handle, bind a stream */
    CHECK_CUBLAS( hipblasCreate(&cublasH) );

    /* step 2: copy data to device */
    CHECK_CUDA( hipMalloc(&d_C, sizeof(half) * batch * m * n) );

    /* step 3: compute */
    CHECK_CUBLAS( hipblasHgemmStridedBatched(cublasH, transa, transb, m, n, k, &alpha, d_A, lda, strideA,
                                            d_B, ldb, strideB, &beta, d_C, ldc, strideC, batch) );

    CHECK_CUBLAS( hipblasScalEx(cublasH, batch * m * n, &scale, HIP_R_32F, d_C, HIP_R_16F, 1, HIP_R_32F) )

    // transpose
    dim3 grid(batch / 32 + 1, m, n);
    transpose_batches<<<grid, 32>>>(d_C, output, batch, m, n);

    /* step 4: copy data to host */
    //CHECK_CUDA( hipMemcpyAsync(output, d_C, sizeof(half) * m * n, hipMemcpyDeviceToDevice, stream));

    /* free resources */
    CHECK_CUDA( hipFree(d_C) );
    CHECK_CUBLAS( hipblasDestroy(cublasH) );
}