#include "hip/hip_runtime.h"
//
// Created by dbettkk on 2022/8/14.
//

#include "cusparse_interface.cuh"

void cusparse_gemm_csr_device(half *sp_A, half *d_B, int m, int k, int n, half *output) {
    float alpha = 1.0f, beta = 0.0f;

    hipsparseHandle_t handle = nullptr;
    CHECK_CUSPARSE(hipsparseCreate(&handle) )

    int ldA = k, ldB = n, ldC = n;
    half *dA_csr_values, *dC;
    int *dA_csr_offsets, *dA_csr_columns;

    CHECK_CUDA( hipMalloc((void **)&dA_csr_offsets, sizeof(int) * (m + 1)) )
    CHECK_CUDA( hipMalloc((void **)&dC, sizeof(half) * m * n) )

    hipsparseDnMatDescr_t matA, matB, matC;
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matA, m, k, ldA, sp_A, HIP_R_16F, HIPSPARSE_ORDER_ROW) )
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, k, n, ldB, d_B, HIP_R_16F, HIPSPARSE_ORDER_ROW) )
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matC, m, n, ldC, dC, HIP_R_16F, HIPSPARSE_ORDER_ROW) )

    hipsparseSpMatDescr_t matA_cmpr;
    // 此时相关参数都未设置
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA_cmpr, m, k, 0, dA_csr_offsets,
                                      nullptr, nullptr, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_16F) )

    void* dBuffer = nullptr;
    size_t bufferSize = 0;
    CHECK_CUSPARSE( hipsparseDenseToSparse_bufferSize(handle, matA, matA_cmpr, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    CHECK_CUSPARSE( hipsparseDenseToSparse_analysis(handle, matA, matA_cmpr, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, dBuffer) )
    // analysis后可以获取相应指针
    int64_t num_rows_tmp, num_cols_tmp, nnz;
    CHECK_CUSPARSE( hipsparseSpMatGetSize(matA_cmpr, &num_rows_tmp, &num_cols_tmp, &nnz) )

    // allocate CSR column indices and values
    CHECK_CUDA( hipMalloc((void**) &dA_csr_columns, nnz * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dA_csr_values,  nnz * sizeof(half)) )
    // reset offsets, column indices, and values pointers
    CHECK_CUSPARSE( hipsparseCsrSetPointers(matA_cmpr, dA_csr_offsets, dA_csr_columns, dA_csr_values) )

    CHECK_CUSPARSE( hipsparseDenseToSparse_convert(handle, matA, matA_cmpr, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, dBuffer) )

    // calculate
    // 当 A/B/C 都为 HIP_R_16F，computeType 需要为 HIP_R_32F
    CHECK_CUSPARSE( hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                            &alpha, matA_cmpr, matB, &beta, matC, HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    CHECK_CUSPARSE( hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha,
                                 matA_cmpr, matB, &beta, matC, HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer) )

    CHECK_CUDA( hipMemcpy(output, dC, sizeof(half) * m * n, hipMemcpyDeviceToHost) )

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA_cmpr) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )

    CHECK_CUDA( hipFree(dBuffer) )
    CHECK_CUDA( hipFree(dA_csr_offsets) )
    CHECK_CUDA( hipFree(dA_csr_columns) )
    CHECK_CUDA( hipFree(dA_csr_values) )
}

void cusparse_gemm_blocked_device_test() {
    int   A_num_rows      = 4;
    int   A_num_cols      = 4;
    int   A_ell_blocksize = 2;
    int   A_ell_cols      = 2;
    int   A_num_blocks    = A_ell_cols * A_num_rows /
                            (A_ell_blocksize * A_ell_blocksize);
    int   B_num_rows      = A_num_cols;
    int   B_num_cols      = 3;
    int   ldb             = B_num_rows;
    int   ldc             = A_num_rows;
    int   B_size          = ldb * B_num_cols;
    int   C_size          = ldc * B_num_cols;
    int   hA_columns[]    = { 1, 0};
    __half hA_values[]    = { 1.0f, 2.0f, 3.0f, 4.0f,
                              5.0f, 6.0f, 7.0f, 8.0f};
    __half hB[]           = { 1.0f,  2.0f,  3.0f,  4.0f,
                              5.0f,  6.0f,  7.0f,  8.0f,
                              9.0f, 10.0f, 11.0f, 12.0f };
    __half hC[]           = { 0.0f, 0.0f, 0.0f, 0.0f,
                              0.0f, 0.0f, 0.0f, 0.0f,
                              0.0f, 0.0f, 0.0f, 0.0f };
    __half hC_result[]    = { 11.0f, 25.0f,  17.0f,  23.0f,
                              23.0f, 53.0f,  61.0f,  83.0f,
                              35.0f, 81.0f, 105.0f, 143.0f };
    float alpha           = 1.0f;
    float beta            = 0.0f;
    //--------------------------------------------------------------------------
    // Check compute capability
    hipDeviceProp_t props{};
    CHECK_CUDA( hipGetDeviceProperties(&props, 0) )
    if (props.major < 7) {
        std::printf("hipsparseSpMM with blocked ELL format is supported only "
                    "with compute capability at least 7.0\n");
        return ;
    }
    //--------------------------------------------------------------------------
    // Device memory management
    int    *dA_columns;
    __half *dA_values, *dB, *dC;
    CHECK_CUDA( hipMalloc((void**) &dA_columns, A_num_blocks * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dA_values,
                           A_ell_cols * A_num_rows * sizeof(__half)) )
    CHECK_CUDA( hipMalloc((void**) &dB, B_size * sizeof(__half)) )
    CHECK_CUDA( hipMalloc((void**) &dC, C_size * sizeof(__half)) )

    CHECK_CUDA( hipMemcpy(dA_columns, hA_columns,
                           A_num_blocks * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_values, hA_values,
                           A_ell_cols * A_num_rows * sizeof(__half),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dB, hB, B_size * sizeof(__half),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dC, hC, C_size * sizeof(__half),
                           hipMemcpyHostToDevice) )
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix A in blocked ELL format
//    CHECK_CUSPARSE( hipsparseCreateBlockedEll(
//            &matA,
//            A_num_rows, A_num_cols, A_ell_blocksize,
//            A_ell_cols, dA_columns, dA_values,
//            HIPSPARSE_INDEX_32I,
//            HIPSPARSE_INDEX_BASE_ZERO, HIP_R_16F) )
    // Create dense matrix B
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, A_num_cols, B_num_cols, ldb, dB,
                                        HIP_R_16F, HIPSPARSE_ORDER_COL) )
    // Create dense matrix C
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matC, A_num_rows, B_num_cols, ldc, dC,
                                        HIP_R_16F, HIPSPARSE_ORDER_COL) )
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMM_bufferSize(
            handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, matA, matB, &beta, matC, HIP_R_32F,
            HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    auto tt = new CudaTime();
    tt->initAndStart();
    // execute SpMM
    CHECK_CUSPARSE( hipsparseSpMM(handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer) )
    printf("time: %fms\n", tt->endAndGetTime());

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    //--------------------------------------------------------------------------
    // device result check
    CHECK_CUDA( hipMemcpy(hC, dC, C_size * sizeof(__half),
                           hipMemcpyDeviceToHost) )
    //int correct = 1;
    for (int i = 0; i < A_num_rows; i++) {
        for (int j = 0; j < B_num_cols; j++) {
            float c_value  = static_cast<float>(hC[i + j * ldc]);
            float c_result = static_cast<float>(hC_result[i + j * ldc]);
            if (c_value != c_result) {
                //correct = 0; // direct floating point comparison is not reliable
                break;
            }
        }
    }

    //--------------------------------------------------------------------------
    // device memory deallocation
    CHECK_CUDA( hipFree(dBuffer) )
    CHECK_CUDA( hipFree(dA_columns) )
    CHECK_CUDA( hipFree(dA_values) )
    CHECK_CUDA( hipFree(dB) )
    CHECK_CUDA( hipFree(dC) )
}
void cusparse_gemm_csr_device_test() {
    int   A_num_rows      = 4;
    int   A_num_cols      = 4;
    int   A_nnz           = 9;
    int   B_num_rows      = A_num_cols;
    int   B_num_cols      = 3;
    int   ldb             = B_num_rows;
    int   ldc             = A_num_rows;
    int   B_size          = ldb * B_num_cols;
    int   C_size          = ldc * B_num_cols;
    int   hA_csrOffsets[] = { 0, 3, 4, 7, 9 };
    int   hA_columns[]    = { 0, 2, 3, 1, 0, 2, 3, 1, 3 };
    float hA_values[]     = { 1.0f, 2.0f, 3.0f, 4.0f, 5.0f,
                              6.0f, 7.0f, 8.0f, 9.0f };
    float hB[]            = { 1.0f,  2.0f,  3.0f,  4.0f,
                              5.0f,  6.0f,  7.0f,  8.0f,
                              9.0f, 10.0f, 11.0f, 12.0f };
    float hC[]            = { 0.0f, 0.0f, 0.0f, 0.0f,
                              0.0f, 0.0f, 0.0f, 0.0f,
                              0.0f, 0.0f, 0.0f, 0.0f };
    float hC_result[]     = { 19.0f,  8.0f,  51.0f,  52.0f,
                              43.0f, 24.0f, 123.0f, 120.0f,
                              67.0f, 40.0f, 195.0f, 188.0f };
    float alpha           = 1.0f;
    float beta            = 0.0f;
    //--------------------------------------------------------------------------
    // Device memory management
    int   *dA_csrOffsets, *dA_columns;
    float *dA_values, *dB, *dC;
    CHECK_CUDA( hipMalloc((void**) &dA_csrOffsets,
                           (A_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dA_columns, A_nnz * sizeof(int))    )
    CHECK_CUDA( hipMalloc((void**) &dA_values,  A_nnz * sizeof(float))  )
    CHECK_CUDA( hipMalloc((void**) &dB,         B_size * sizeof(float)) )
    CHECK_CUDA( hipMalloc((void**) &dC,         C_size * sizeof(float)) )

    CHECK_CUDA( hipMemcpy(dA_csrOffsets, hA_csrOffsets,
                           (A_num_rows + 1) * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_columns, hA_columns, A_nnz * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_values, hA_values, A_nnz * sizeof(float),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dB, hB, B_size * sizeof(float),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dC, hC, C_size * sizeof(float),
                           hipMemcpyHostToDevice) )
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_nnz,
                                      dA_csrOffsets, dA_columns, dA_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    // Create dense matrix B
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, A_num_cols, B_num_cols, ldb, dB,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL) )
    // Create dense matrix C
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matC, A_num_rows, B_num_cols, ldc, dC,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL) )
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMM_bufferSize(
            handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, matA, matB, &beta, matC, HIP_R_32F,
            HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute SpMM
    auto tt = new CudaTime();
    tt->initAndStart();
    CHECK_CUSPARSE( hipsparseSpMM(handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer) )
    printf("time: %fms\n", tt->endAndGetTime());
    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    //--------------------------------------------------------------------------
    // device result check
    CHECK_CUDA( hipMemcpy(hC, dC, C_size * sizeof(float),
                           hipMemcpyDeviceToHost) )
    //int correct = 1;
    for (int i = 0; i < A_num_rows; i++) {
        for (int j = 0; j < B_num_cols; j++) {
            if (hC[i + j * ldc] != hC_result[i + j * ldc]) {
                //correct = 0; // direct floating point comparison is not reliable
                break;
            }
        }
    }
    //--------------------------------------------------------------------------
    // device memory deallocation
    CHECK_CUDA( hipFree(dBuffer) )
    CHECK_CUDA( hipFree(dA_csrOffsets) )
    CHECK_CUDA( hipFree(dA_columns) )
    CHECK_CUDA( hipFree(dA_values) )
    CHECK_CUDA( hipFree(dB) )
    CHECK_CUDA( hipFree(dC) )
}