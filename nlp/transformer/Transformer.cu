#include "hip/hip_runtime.h"
//
// Created by dbettkk on 2022/8/1.
//

#include "Transformer.cuh"

void Transformer::PositionalEncoding(MatrixHalf *in, MatrixHalf *out) {
    in->addMatrix(pe, out);
}

void Transformer::make_pe(int max_len, int d_model, MatrixHalf *out) {
    auto div_term = new double[d_model / 2 + 1];
    for (int i = 0; i < d_model / 2 + 1; i++) {
        div_term[i] = exp(i * 2 * -log(10000.0) / d_model);
    }
    half *pe = new half[max_len * d_model];
    for (int i = 0; i < max_len; i++) {
        for (int j = 0; j < d_model; j++) {
            int idx = i * d_model + j;
            if (j % 2 == 0) {
                pe[idx] = sin(div_term[j / 2]);
            } else {
                pe[idx] = cos(div_term[j / 2]);
            }
        }
    }
    half *d_pe;
    hipMalloc(&d_pe, sizeof(half) * max_len * d_model);
    hipMemcpy(d_pe, pe, sizeof(half) * max_len * d_model, hipMemcpyHostToDevice);
    out->setMatrix(d_pe);

    delete[] div_term;
    delete[] pe;
}

void Transformer::init(int max_len, int d_model) {
    pe = new MatrixHalf(1, max_len, d_model, true);
    make_pe(max_len, d_model, pe);
}

void Transformer::make_mask1(int max_len, MatrixHalf *out) {
    // 从主对角线开始 隔两个对角线的值不mask
    half *h_mask = new half[max_len * max_len];
    memset(h_mask, 0, sizeof(half) * max_len * max_len);
    int max_num = (max_len - 1) / 3;
    for (int i = 0; i < max_len; i++) {
        for (int j = 0; j < max_len; j++) {
            for (int k = 0; k <= max_num; k++) {
                if (i == j + k * 3) h_mask[i * max_len + j] = 1;
                if (j == i + k * 3) h_mask[i * max_len + j] = 1;
            }
        }
    }
    for (int i = 0; i < max_len; i++) {
        for (int j = 0; j < max_len; j++) {
            printf("%d ", __half2int_rz(h_mask[i * max_len + j]));
        }
        printf("\n");
    }
}
